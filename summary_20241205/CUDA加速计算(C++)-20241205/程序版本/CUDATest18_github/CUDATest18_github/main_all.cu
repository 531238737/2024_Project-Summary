#include "hip/hip_runtime.h"
﻿#include <cmath> 

#include <iostream>
#include <Eigen/Dense>
#include<vector>
#include <fstream>
#include <sstream>
#include<string>
#include<ctime>
#include<time.h>
#include <chrono>
#include <algorithm>
#include<opencv2/opencv.hpp>
#include <unsupported/Eigen/CXX11/Tensor>
#include <cmath> 
#include <hip/hip_runtime.h>

using namespace cv;
using namespace std;
using namespace Eigen;
#include "mat.h"
#include"hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"paraeters_all.h"



//以下代码作用是屏蔽waring警告
#pragma warning(default : 1234)
#pragma warning(disable : 65)

#pragma   warning(   disable  :   4290   ) 
#pragma warning(disable : 266)
#pragma warning(disable : 29)
#pragma warning(disable : 20
#pragma warning(disable : 65)
#pragma warning(disable : 169)










/// <summary>
/// 核函数，用来计算TFM成像区域
/// </summary>
/// <param name="temp_sum_image">最终成像矩阵</param>
/// <param name="RData">FMC数据</param>
/// <param name="time_ruler_st_all">延迟时间矩阵</param>
/// <param name="rows">成像区域的行数</param>
/// <param name="cols">成像区域的列数</param>
/// <param name="depth">source数</param>
/// <param name="RData_rows">FMC数据的行数</param>
/// <returns></returns>
__global__ void calculateDistances_all_8(int* temp_sum_image,int16_t* RData, int16_t* time_ruler_st_all, int rows, int cols, int depth, int RData_rows) {
    int k = blockIdx.x;//小矩阵在大矩阵中的x索引
    int l = blockIdx.y;//小矩阵在大矩阵中的y索引

    int tx = threadIdx.x;//像素点在其所在小矩阵中的x索引
    int ty = threadIdx.y;//像素点在其所在小矩阵中的y索引
    int tid = ty * blockDim.x + tx;//像素带你在其所在小矩阵中的位置索引
    int threads_per_block = blockDim.x * blockDim.y;//每个block包含的线程数

    int local_sum = 0;

    //计算像素点
    for (int idx = tid; idx < depth * depth; idx += threads_per_block) {
        int i = idx / depth;
        int j = idx % depth;
        local_sum += RData[time_ruler_st_all[(j * cols + l) * rows * depth + (i * rows + k)] +
            RData_rows * (j * depth + i)];
    }

    // 使用warp-level primitives进行归约
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        local_sum += __shfl_down_sync(0xFFFFFFFF, local_sum, offset);
    }

    // 块内的第一个线程将warp-level归约的结果写入共享内存
    extern __shared__ int shared_sums[];
    if (tid % warpSize == 0) {
        shared_sums[tid / warpSize] = local_sum;
    }
    __syncthreads();

    // 归约warp结果
    if (tid == 0) {
        int block_sum = 0;
        for (int i = 0; i < blockDim.x * blockDim.y / warpSize; ++i) {
            block_sum += shared_sums[i];
        }
        temp_sum_image[l * rows + k] = block_sum;
    }
}








/// <summary>
/// vector变量转Matrix变量
/// </summary>
/// <param name="data"></param>
/// <returns></returns>
MatrixXd vector2DToMatrix(const vector<vector<double>>& data) {
    int rows = data.size();
    int cols = data[0].size();

    MatrixXd matrix = Map<const MatrixXd>(data[0].data(), rows, cols).eval();

    return matrix;
}







/// <summary>
/// 从 CSV 文件中读取数据并保存为vector矩阵
/// </summary>
/// <param name="filename">文件名</param>
/// <returns>vector矩阵</returns>
vector<vector<int>> readCSVToMatrix(const string& filename) {
    ifstream file(filename);
    string line;
    vector<vector<int>> matrix;

    if (file.is_open()) {
        while (getline(file, line)) {
            istringstream sline(line);
            string field;
            vector<int> row;

            while (getline(sline, field, ',')) {
                row.push_back(stoi(field)); // 将字符串转换为整数并添加到行中
            }

            matrix.push_back(row); // 将行添加到矩阵中
        }

        file.close();

    }

    return matrix;
}





//读取csv文件
template<typename M>
M load_csv(const string& path) {
    try {

        ifstream indata;
        indata.open(path);
        if (!indata.is_open()) {
            throw runtime_error("Error opening file");
        }

        string line;
        vector<double> values;
        unsigned int rows = 0;
        while (getline(indata, line)) {
            stringstream lineStream(line);
            string cell;
            while (getline(lineStream, cell, ',')) {
                values.push_back(stod(cell));
            }
            ++rows;
        }

        indata.close();


        return Map<const Matrix<typename M::Scalar, M::RowsAtCompileTime, M::ColsAtCompileTime, RowMajor>>(values.data(), rows, values.size() / rows);
    }
    catch (const exception& e) {
        cerr << "Exception caught: " << e.what() << endl;

        throw;
    }
}



/// <summary>
/// 读取csv文件转换为MatrixXd矩阵
/// </summary>
/// <param name="path">路径</param>
/// <returns>MatrixXd矩阵</returns>
MatrixXd read_csv(const string& path) {
    ifstream file(path);
    MatrixXd out(1500, 128);
    if (!file.is_open()) {
        throw runtime_error("Error opening file");

    }
    stringstream buffer;
    buffer << file.rdbuf();
    string cell;
    string line;
    int i = 0;
    while (getline(buffer, line, '\n')) {
        stringstream lineStream(line);
        string cell;
        int j = 0;
        while (getline(lineStream, cell, ',')) {
            out(i, j) = stod(cell);
            j++;
        }
        i++;
    }
    return out;

}
/// <summary>
/// 读取mat文件
/// </summary>
/// <param name="filePath">路径</param>
/// <param name="matrixName">矩阵名字</param>
/// <param name="RData">FMC矩阵</param>
void ReadMatlabMat(string  filePath, string matrixName, MatrixXd& RData)
{
    MATFile* pmatFile = NULL;
    mxArray* pMxArray = NULL;
    double* matdata;

    pmatFile = matOpen(filePath.c_str(), "r");//打开.mat文件
    if (pmatFile == NULL)
    {
        cout << "打开文件失败" << endl;
        return;
    }
    // assert(pmatFile != NULL);
    pMxArray = matGetVariable(pmatFile, matrixName.c_str());//获取.mat文件里面名为matrixName的矩阵

    matdata = mxGetPr(pMxArray);
    int rows = mxGetM(pMxArray); // 获取矩阵的行数
    int cols = mxGetN(pMxArray); // 获取矩阵的列数



    matClose(pmatFile);//close file



    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {

            RData(i, j) = matdata[j * rows + i];
        }
    }
    mxDestroyArray(pMxArray);//释放内存
    matdata = NULL;
    return;
}



/// <summary>
/// 读取mat文件并转换为32位有符号整型
/// </summary>
/// <param name="filePath">路径</param>
/// <param name="matrixName">矩阵名称</param>
/// <param name="input">FMC矩阵</param>
void ReadMatlabMat(string  filePath, string matrixName, MatrixXi& input)
{
    MATFile* pmatFile = NULL;
    mxArray* pMxArray = NULL;



    pmatFile = matOpen(filePath.c_str(), "r");//打开.mat文件
    if (pmatFile == NULL)
    {
        cout << "打开文件失败" << endl;
        return;
    }
    else
    {
        cout << "打开文件成功" << endl;

    }

    // assert(pmatFile != NULL);
    pMxArray = matGetVariable(pmatFile, matrixName.c_str());//获取.mat文件里面名为matrixName的矩阵



    int32_T* pData = (int32_T*)mxGetData(pMxArray); // 获取整数数据指针
    // 进行相关操作



    int rows = mxGetM(pMxArray); // 获取矩阵的行数
    int cols = mxGetN(pMxArray); // 获取矩阵的列数



    matClose(pmatFile);//close file



    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {

            input(i, j) = (pData[j * rows + i]);
        }
    }
    mxDestroyArray(pMxArray);//释放内存

    return;
}


/// <summary>
/// 获取mat文件矩阵指针
/// </summary>
/// <param name="filePath">路径</param>
/// <param name="matrixName">矩阵名字</param>
/// <param name="time_ruler_st_pointer">延迟时间矩阵指针</param>
/// <param name="pmatFile">指向 MATFile 类型的指针，用于处理 .mat 文件的文件对象。</param>
/// <param name="pMxArray">指向 mxArray 类型的指针，用于存储从 .mat 文件中读取的矩阵数据</param>
void ReadMatlabDataAndProcess(string filePath, string matrixName, int32_T*& time_ruler_st_pointer, MATFile*& pmatFile, mxArray*& pMxArray) {

    pmatFile = matOpen(filePath.c_str(), "r");//打开.mat文件
    if (pmatFile == NULL)
    {
        cout << "打开文件失败" << endl;
        return;
    }
    else
    {
        cout << "打开文件成功" << endl;

    }

    // assert(pmatFile != NULL);
    pMxArray = matGetVariable(pmatFile, matrixName.c_str());//获取.mat文件里面名为matrixName的矩阵
    if (pMxArray == NULL) {
        cout << "获取矩阵失败" << endl;
        matClose(pmatFile);
        return;
    }
    time_ruler_st_pointer = (int32_T*)mxGetData(pMxArray); // 获取整数数据指针

}



/// <summary>
/// 获取mat文件矩阵指针转换为16位整数
/// </summary>
/// <param name="filePath">路径</param>
/// <param name="matrixName">矩阵名字</param>
/// <param name="time_ruler_st_pointer">延迟时间矩阵指针</param>
/// <param name="pmatFile">指向 MATFile 类型的指针，用于处理 .mat 文件的文件对象。</param>
/// <param name="pMxArray">指向 mxArray 类型的指针，用于存储从 .mat 文件中读取的矩阵数据</param>
void ReadMatlabDataAndProcess(string filePath, string matrixName, int16_T*& time_ruler_st_pointer, MATFile*& pmatFile, mxArray*& pMxArray) {

    pmatFile = matOpen(filePath.c_str(), "r");//打开.mat文件
    if (pmatFile == NULL)
    {
        cout << "打开文件失败" << endl;
        return;
    }
    else
    {
        cout << "打开文件成功" << endl;

    }

    // assert(pmatFile != NULL);
    pMxArray = matGetVariable(pmatFile, matrixName.c_str());//获取.mat文件里面名为matrixName的矩阵
    if (pMxArray == NULL) {
        cout << "获取矩阵失败" << endl;
        matClose(pmatFile);
        return;
    }
    time_ruler_st_pointer = (int16_T*)mxGetData(pMxArray); // 获取整数数据指针

}







/// <summary>
/// 获取mat文件矩阵指针转换为32位整数
/// </summary>
/// <param name="filePath">路径</param>
/// <param name="matrixName">矩阵名字</param>
/// <param name="time_ruler_st_pointer">延迟时间矩阵指针</param>
/// <param name="pmatFile">指向 MATFile 类型的指针，用于处理 .mat 文件的文件对象。</param>
/// <param name="pMxArray">指向 mxArray 类型的指针，用于存储从 .mat 文件中读取的矩阵数据</param>
MatrixXi ReadMatlabMat(string  filePath, string matrixName)
{
    MATFile* pmatFile = NULL;
    mxArray* pMxArray = NULL;
    MatrixXi out(1500, 128);


    pmatFile = matOpen(filePath.c_str(), "r");//打开.mat文件
    if (pmatFile == NULL)
    {
        cout << "打开文件失败" << endl;
        return out;
    }
    // assert(pmatFile != NULL);
    pMxArray = matGetVariable(pmatFile, matrixName.c_str());//获取.mat文件里面名为matrixName的矩阵



    int32_T* pData = (int32_T*)mxGetData(pMxArray); // 获取整数数据指针
    // 进行相关操作



    int rows = mxGetM(pMxArray); // 获取矩阵的行数
    int cols = mxGetN(pMxArray); // 获取矩阵的列数




    matClose(pmatFile);//close file



    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {

            out(i, j) = (pData[j * rows + i]);
        }
    }
    mxDestroyArray(pMxArray);//释放内存

    return out;
}







/// <summary>
/// 保存数据成csv
/// </summary>
/// <param name="fileName">文件名</param>
/// <param name="matrix">保存MatrixXd矩阵</param>
void saveData(string fileName, MatrixXd  matrix)
{

    const static IOFormat CSVFormat(FullPrecision, DontAlignCols, ", ", "\n");

    ofstream file(fileName);
    if (file.is_open())
    {
        file << matrix.format(CSVFormat);
        file.close();
    }
}

/// <summary>
/// 保存数据成csv
/// </summary>
/// <param name="fileName">文件名</param>
/// <param name="matrix">保存MatrixXi矩阵</param>
void saveData(string fileName, MatrixXi  matrix)
{

    const static IOFormat CSVFormat(FullPrecision, DontAlignCols, ", ", "\n");

    ofstream file(fileName);
    if (file.is_open())
    {
        file << matrix.format(CSVFormat);
        file.close();
    }
}
/// <summary>
/// 保存数据成csv
/// </summary>
/// <param name="fileName">文件名</param>
/// <param name="matrix">保存RowVectorXd矩阵</param>
void saveData(string fileName, RowVectorXd  matrix)
{

    const static IOFormat CSVFormat(FullPrecision, DontAlignCols, ", ", "\n");

    ofstream file(fileName);
    if (file.is_open())
    {
        file << matrix.format(CSVFormat);
        file.close();
    }
}
/// <summary>
/// 保存数据成csv
/// </summary>
/// <param name="fileName">文件名</param>
/// <param name="matrix">保存RowVectorXi矩阵</param>
void saveData(string fileName, RowVectorXi  matrix)
{

    const static IOFormat CSVFormat(FullPrecision, DontAlignCols, ", ", "\n");

    ofstream file(fileName);
    if (file.is_open())
    {
        file << matrix.format(CSVFormat);
        file.close();
    }
}


/// <summary>
/// 渲染图像
/// </summary>
/// <param name="matrix">图像矩阵</param>
void scaleMatrixToImage(const MatrixXd& matrix) {
    // 找到矩阵中的最小值和最大值
    double minVal = matrix.minCoeff();
    double maxVal = matrix.maxCoeff();

    // 创建一个 CV_8U 类型的图像矩阵
    Mat image(matrix.rows(), matrix.cols(), CV_8U);

    // 缩放矩阵数据到 [0, 255] 范围
    /*MatrixXd scaledMatrix = (matrix.array() - minVal) / (maxVal - minVal) * 255.0;*/
    MatrixXd scaledMatrix = (matrix.array() - minVal) / (maxVal - minVal) * 255.0;
    scaledMatrix = scaledMatrix.array().max(0.0).min(255.0); // 将数据限制在 [0, 255] 范围内
    // 将矩阵数据转换为 CV_8U 类型的图像数据
    for (int i = 0; i < matrix.rows(); i++) {
        for (int j = 0; j < matrix.cols(); j++) {
            image.at<uchar>(i, j) = static_cast<uchar>(scaledMatrix(i, j));
        }
    }

    // 创建带有外边距的新图像


    applyColorMap(image, image, COLORMAP_JET);

    // 显示图像
    namedWindow("Scaled Image", WINDOW_NORMAL);
    imshow("Scaled Image", image);
    waitKey(0);


}


/// <summary>
/// 渲染图象
/// </summary>
/// <param name="matrix">整数图像矩阵</param>
void scaleMatrixToImage(const MatrixXi& matrix) {
    // 找到矩阵中的最小值和最大值
    int minVal = matrix.minCoeff();
    int maxVal = matrix.maxCoeff();

    // 创建一个 CV_8U 类型的图像矩阵
    Mat image(matrix.rows(), matrix.cols(), CV_8U);

    // 缩放矩阵数据到 [0, 255] 范围

    MatrixXd scaledMatrix = (matrix.cast<double>().array() - minVal) / (maxVal - minVal) * 255.0;
    // saveData("out/mat/scaledMatrix1.csv", scaledMatrix);
    scaledMatrix = scaledMatrix.array().max(0.0).min(255.0); // 将数据限制在 [0, 255] 范围内
     saveData("out/mat/scaledMatrix_all.csv", scaledMatrix);
     // 将矩阵数据转换为 CV_8U 类型的图像数据
    for (int i = 0; i < matrix.rows(); i++) {
        for (int j = 0; j < matrix.cols(); j++) {
            image.at<uchar>(i, j) = static_cast<uchar>(scaledMatrix(i, j));
        }
    }

    // 创建带有外边距的新图像


    applyColorMap(image, image, COLORMAP_JET);
    cout << "显示图像" << endl;
    string winname = "MyWindow";


    // 显示图像
    namedWindow(winname, WINDOW_KEEPRATIO);
    imshow(winname, image);
    waitKey(0);



}







/// <summary>
/// 主函数
/// </summary>
/// <returns></returns>
MatrixXi cu_main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);//获取GPU信息
    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        cout << "使用GPU device " << i << ": " << devProp.name << endl;
        cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        cout << "SM的数量：" << devProp.multiProcessorCount << endl;
        cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << endl;
        cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << endl;
        cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << endl;

        cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << endl;
        cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << endl;
        cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << endl;
        cout << "======================================================" << endl;

    }

    MatrixXi  temp_sum_image = MatrixXi::Zero(ny, nx);


    try {

        auto start = chrono::high_resolution_clock::now();
        //初始化各项参数
        //申请并分配内存

        int* d_temp_sum_image;
        int16_T* d_mat_time_ruler_st, * time_ruler_st_pointer, * d_mat_RData, * RData_pointer;//申明指针


        auto start1 = chrono::high_resolution_clock::now();
        string filename1 = "data/mat/RData_int16.mat";

        MATFile* pmatFile_RData = NULL;
        mxArray* pMxArray_RData = NULL;
        MATFile* pmatFile = NULL;
        mxArray* pMxArray = NULL;
        string matrixName1 = "result";
        //ReadMatlabMat(filename, "result", RData);
        ReadMatlabDataAndProcess(filename1, matrixName1, RData_pointer, pmatFile_RData, pMxArray_RData);//读取FMC数据


        string filename2 = "data/mat/time_ruler_st/time_ruler_st_mini.mat";
        string matrixName2 = "result_some";
        ReadMatlabDataAndProcess(filename2, matrixName2, time_ruler_st_pointer, pmatFile, pMxArray);//读取延迟时间矩阵

        //位gpu分配内存
        hipMalloc(&d_temp_sum_image, nx_ny_int);
        hipMalloc(&d_mat_RData, RData_int16);
        hipMalloc(&d_mat_time_ruler_st, mat_time_ruler_st_int16);


        // 分配数据
        hipMemcpy(d_temp_sum_image, temp_sum_image.data(), nx_ny_int, hipMemcpyHostToDevice);
        hipMemcpy(d_mat_RData, RData_pointer, RData_int16, hipMemcpyHostToDevice);
        hipMemcpy(d_mat_time_ruler_st, time_ruler_st_pointer, mat_time_ruler_st_int16, hipMemcpyHostToDevice);
        auto end1 = chrono::high_resolution_clock::now();
        // 计算时间差
        chrono::duration<double> duration1 = end1 - start1;
        cout << "读取RData与time_ruler_st花费时间：" << duration1.count() << "s" << endl;


        int RData_rows = 1500;



        auto start2 = chrono::high_resolution_clock::now();










        // 定义线程块大小
        dim3 blockDim(32, 32);  // 16x16 = 256 个线程

        // 定义网格大小
        dim3 gridDim(80, 130);  // 80 个 k 值，130 个 l 值

        // 计算共享内存大小（以字节为单位）
        size_t shared_mem_size = 1024 * sizeof(int32_t);  // 每个线程一个 int

        

        // 启动核函数
        calculateDistances_all_8 << <gridDim, blockDim, shared_mem_size >> > (d_temp_sum_image, d_mat_RData, d_mat_time_ruler_st, ny, nx, nrec,  RData_rows);


        hipDeviceSynchronize();//等待gpu任务完成
        auto end2 = chrono::high_resolution_clock::now();

        chrono::duration<double> duration2 = end2 - start2;
        cout << "cuda运行花费时间" << duration2.count() << "秒" << endl;

        ;
        hipMemcpy(temp_sum_image.data(), d_temp_sum_image, nx_ny_int, hipMemcpyDeviceToHost);


        //释放gpu内存
        hipFree(d_temp_sum_image);

        hipFree(d_mat_time_ruler_st);
        hipFree(d_mat_RData);

        matClose(pmatFile);
        mxDestroyArray(pMxArray);//释放内存
        matClose(pmatFile_RData);
        mxDestroyArray(pMxArray_RData);//释放内存
        // 最终保存整个计算过程中的temp_sum_image
        //saveData("out/mat/temp_sum_image_final_mat.csv", temp_sum_image);
        hipDeviceReset();//重置CUDA，释放所有内存，销毁事件
        auto end = chrono::high_resolution_clock::now();
        // 计算时间差
        chrono::duration<double> duration = end - start;
        // 输出程序运行时间，以秒为单位

        cout << "程序运行时间: " << duration.count() << " 秒" << endl;

        saveData("out/mat/scaledMatrix4.csv", temp_sum_image);
        scaleMatrixToImage(temp_sum_image);

        // matrixToImage(temp_sum_image);
    }
    catch (const cv::Exception& e) {
        // 捕获异常并输出异常信息
        cerr << "OpenCV Exception caught: " << e.what() << endl;
    }

    catch (...) {
        // 捕获未知异常并输出异常信息
        cerr << "Unknown Exception caught." << endl;
    }

    return temp_sum_image;
}


